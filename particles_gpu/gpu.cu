#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <vector>
#include <algorithm>


#define NUM_THREADS 256
//#define _cutoff 0.01    //Value copied from common.cpp
//#define _density 0.0005
#define MAXITEM 4 //Assume at most MAXITEM particles in one bin. Change depends on binSize


extern double size;

double binSize;
int binNum;
//
//  benchmarking program
//


__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t * particles, int n)
{
  // Get thread (particle) ID
  __shared__ int a[1024*1024*1024];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particles[tid].ax = particles[tid].ay = 0;
  for(int j = 0 ; j < n ; j++)
    a[j]++;
    //apply_force_gpu(particles[tid], particles[j]);

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

__global__ void getCount(particle_t* particles, int* count,int n,double binSize,int binNum)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = gridDim.x*blockDim.x;
    for(int i=tid;i<n;i+=offset)
    {
        int x = int(particles[i].x / binSize);
        int y = int(particles[i].y / binSize);
        atomicAdd(count+x*binNum+y,1);
    }
}

__global__ void buildBins(particle_t* particles,particle_t* tmp,int* count,int n,double binSize,int binNum)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = gridDim.x*blockDim.x;
    for(int i=tid;i<n;i+=offset)
    {
        int x = int(particles[i].x / binSize);
        int y = int(particles[i].y / binSize);
        int id = atomicSub(count+x*binNum+y,1);
        tmp[id-1] = particles[i];
    }
}

int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles,*tmp;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));
    hipMalloc((void **) &tmp, n * sizeof(particle_t));
    
    std::vector<thrust::device_vector<int> > H(4);
    set_size( n );
    binSize = cutoff*4;  
    binNum = int(size / binSize)+1; // Should be around sqrt(N/2)
    printf("Grid Size: %.4lf\n",size);
    printf("Number of Bins: %d*%d\n",binNum,binNum);
    printf("Bin Size: %.2lf\n",binSize);
    
    
    int* cnt;
    hipMalloc((void **) &cnt, (binNum*binNum+1) * sizeof(int));
    hipMemset(cnt,0,(binNum*binNum+1)*sizeof(int));
    cnt+=1; //Add one therefore cnt[-1]==0
    int* count = (int*) malloc(binNum*binNum * sizeof(int));
    	
    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //
        int threadNum = 512;
        int blockNum = min(512,(n+threadNum-1)/threadNum);
        
        
        //Old methods that don't assume maxitems for each bin
        hipMemset(cnt,0,binNum*binNum*sizeof(int));
        getCount<<<blockNum,threadNum>>>(d_particles,cnt,n,binSize,binNum);
        
        hipMemcpy(count, cnt, binNum*binNum * sizeof(int), hipMemcpyDeviceToHost);
        for(int i=1;i<binNum*binNum;i++)  //Prefix sum  could be faster using parallel....
            count[i]+=count[i-1];
        hipMemcpy(cnt, count, binNum*binNum * sizeof(int), hipMemcpyHostToDevice);
        buildBins<<<blockNum,threadNum>>>(d_particles,tmp,cnt,n,binSize,binNum);
        std::swap(d_particles,tmp);
        hipMemcpy(cnt, count, binNum*binNum * sizeof(int), hipMemcpyHostToDevice);
        
        //int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    	//compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);
    	compute_forces_gpu<<<blockNum,threadNum>>> (d_particles,cnt,n);
        
        //
        //  move particles
        //
    	//move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
	    }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
